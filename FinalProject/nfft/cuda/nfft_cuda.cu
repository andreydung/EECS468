#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// define the NFFTPlan
#include <stdio.h>
#include <math.h>
#include <string.h>
#include <stdlib.h>

#define NFFT_PRECISION_SINGLE
#define __float128 float
#include "../include/nfft3mp.h"



/*
kernel/nfft/nfft.c:#define BASE(x) CEXP(x)
kernel/nfft/nfft.c:        f[j] += f_hat[k_L] * BASE(-II * omega);
kernel/nfft/nfft.c:        f[j] += f_hat[k_L] * BASE(-II * omega);
kernel/nfft/nfft.c:          f_hat[k_L] += f[j] * BASE(II * omega);
kernel/nfft/nfft.c:          f_hat[k_L] += f[j] * BASE(II * omega);
kernel/nfft/nfft.c:        f_hat[k_L] += f[j] * BASE(II * omega);
kernel/nfft/nfft.c:        f_hat[k_L] += f[j] * BASE(II * omega);
double _Complex cexp(double _Complex z);
*/

void CopyToDevice(void* device, void* host, size_t size) 
{
	hipError_t err = hipMemcpy(device, host, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) 
	{
		printf("Error copy to device: %s \n", hipGetErrorString(err));
		exit(-1);
	}
}

void CopyFromDevice(void* device, void* host, size_t size) 
{
	hipError_t err = hipMemcpy(host, device, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) 
	{
		printf("Error copy from device: %s \n", hipGetErrorString(err));
		exit(-1);
	}
}

void* AllocateDevice(size_t size) 
{
	void* out;
	hipError_t err = hipMalloc(&out, size);	
	if (err != hipSuccess) 
	{
		printf("Error allocating: %s \n", hipGetErrorString(err));
		exit(-1);
	}
	return out;
}

void FreeDevice(void* p) 
{
	hipFree(p);	
}


void Cuda_NFFT_trafo_1d(nfft_plan* plan)
{
}

void Cuda_NFFT_trafo_2d(nfft_plan* plan)
{
}



