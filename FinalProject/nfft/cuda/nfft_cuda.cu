#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>


void CopyToDevice(void* device, void* host, size_t size) {
	hipError_t err = hipMemcpy(device, host, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("Error copy to device: %s \n", hipGetErrorString(err));
		exit(-1);
	}
}

void CopyFromDevice(void* device, void* host, size_t size) {
	hipError_t err = hipMemcpy(host, device, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("Error copy from device: %s \n", hipGetErrorString(err));
		exit(-1);
	};
}

void* AllocateDevice(size_t size) {
	void* out;
	hipError_t err = hipMalloc(&out, size);	
	if (err != hipSuccess) {
		printf("Error allocating: %s \n", hipGetErrorString(err));
		exit(-1);
	};
	return out;
}

void FreeDevice(void* p) {
	hipFree(p);	
}